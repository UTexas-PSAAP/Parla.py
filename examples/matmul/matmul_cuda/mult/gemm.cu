#include "hip/hip_runtime_api.h"
#include "singleton.h"
#include "gemm.h"

void make_handle(int device){
  hipSetDevice(device);
  auto const& handle = knnHandle_t::instance();
  hipDeviceSynchronize();
}


void internal_gemm(int m, int n, int k, const float* A, const float* B, float* C, int device) {
  hipSetDevice(device);
  auto const& handle = knnHandle_t::instance();
  float alpha = 1.0;
  float beta = 0.;
  hipblasSgemm(handle.blas, HIPBLAS_OP_N, HIPBLAS_OP_N, 
        m, n, k, &alpha,
        A, m,
        B, k, &beta,
        C, m);
  hipDeviceSynchronize();
}
